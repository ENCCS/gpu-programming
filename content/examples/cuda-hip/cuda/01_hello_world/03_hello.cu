//
// nvcc 03_hello.cu
// run with the command './a.exe' and './a.exe > out.txt'
//	   to get results from CPU and GPU with different output orders
//

#include <hip/hip_runtime.h>
#include <stdio.h>

void __global__ hello_from_gpu()
{
    printf("\n\n---Hello World from the GPU!\n\n");
}


int main(int argc, const char * argv[])
{
	printf("\n----------------------\n");
	printf("Hello World from CPU! Before calling 'hello_from_gpu' kernel function.\n");
	
    hello_from_gpu<<<1, 1>>>();

	printf("Hello World from CPU!  After calling 'hello_from_gpu' kernel function.\n");
	printf("\n----------------------\n");

	hipDeviceSynchronize(); // cudaDeviceReset();

    return 0;
}
