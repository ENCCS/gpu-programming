//
// nvcc 05_hello.cu
//

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x;
    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;
    printf("Hello World from GPU (block-%d and thread-(%d, %d))!\n", bid, tidx, tidy);
}


int main(int argc, const char * argv[])
{
	printf("\n----------------------\n");
	printf("Hello World from CPU! Before calling 'hello_from_gpu' kernel function.\n");

    const dim3 block_size(4, 8);
    hello_from_gpu<<<1, block_size>>>();

	printf("Hello World from CPU!  After calling 'hello_from_gpu' kernel function.\n");
	printf("\n----------------------\n");

    hipDeviceSynchronize(); // cudaDeviceReset();
    return 0;
}
